
#include <hip/hip_runtime.h>
//
// Created by android on 26.7.2025.
//
// Matrix_Dot.cu

/*
Copyright [2025] [Pirkka Toivakka]

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
// This project uses model weights licensed under CC BY 4.0 (see /Models/LICENSE)

extern "C" {
    __global__ void matmul_kernel(const float* A, const float* B, float* C, int N, int M, int K) {
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        if(row < N && col < K) {
            float sum = 0.0f;
            for(int i = 0; i < M; ++i)
                sum += A[row * M + i] * B[i * K + col];
            C[row * K + col] = sum;
        }
    }

    void matmul(const float* A, const float* B, float* C, int N, int M, int K) {
        float *d_A, *d_B, *d_C;
        size_t size_A = N * M * sizeof(float);
        size_t size_B = M * K * sizeof(float);
        size_t size_C = N * K * sizeof(float);

        hipMalloc(&d_A, size_A);
        hipMalloc(&d_B, size_B);
        hipMalloc(&d_C, size_C);

        hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);

        dim3 block(16, 16);
        dim3 grid((K + 15)/16, (N + 15)/16);

        matmul_kernel<<<grid, block>>>(d_A, d_B, d_C, N, M, K);

        hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

        hipFree(d_A); hipFree(d_B); hipFree(d_C);
    }
}
