
#include <hip/hip_runtime.h>
// Matrix_MPly.cu (CUDA-ytimellä)

__global__ void matmply_kernel(const float* A, const float* B, float* C, int n, int m) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = n * m;
    if (idx < total) {
        C[idx] = A[idx] * B[idx];
    }
}

extern "C" 
void matmply(const float* A, const float* B, float* C, int n, int m) {
    float *d_A, *d_B, *d_C;
    int size = n * m * sizeof(float);
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n * m + blockSize - 1) / blockSize;
    matmply_kernel<<<numBlocks, blockSize>>>(d_A, d_B, d_C, n, m);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
}

