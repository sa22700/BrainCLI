
#include <hip/hip_runtime.h>
// Matrix_Dot.cu (CUDA-ytimellä)

extern "C"
__global__ void matmul_kernel(const float* A, const float* B, float* C, int n, int m, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n && col < k) {
        float sum = 0.0f;
        for (int i = 0; i < m; ++i)
            sum += A[row * m + i] * B[i * k + col];
        C[row * k + col] = sum;
    }
}

extern "C"
void matmul(const float* A, const float* B, float* C, int n, int m, int k) {
    float *d_A, *d_B, *d_C;
    size_t size_A = n * m * sizeof(float);
    size_t size_B = m * k * sizeof(float);
    size_t size_C = n * k * sizeof(float);
    hipMalloc(&d_A, size_A); hipMalloc(&d_B, size_B); hipMalloc(&d_C, size_C);
    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);
    dim3 blockSize(16, 16);
    dim3 numBlocks((k + 15)/16, (n + 15)/16);
    matmul_kernel<<<numBlocks, blockSize>>>(d_A, d_B, d_C, n, m, k);
    hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
}

