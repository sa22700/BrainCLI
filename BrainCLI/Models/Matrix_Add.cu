
#include <hip/hip_runtime.h>
// Matrix_Add.cu (CUDA-ytimellä)

extern "C"
__global__ void addmat_kernel(const float* A, const float* B, float* C, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        C[idx] = A[idx] + B[idx];
    }
}

extern "C"
void addmat(const float* A, const float* B, float* C, int n, int m) {
    int size = n * m;
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size * sizeof(float));
    hipMalloc(&d_B, size * sizeof(float));
    hipMalloc(&d_C, size * sizeof(float));
    hipMemcpy(d_A, A, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size * sizeof(float), hipMemcpyHostToDevice);
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    addmat_kernel<<<numBlocks, blockSize>>>(d_A, d_B, d_C, size);
    hipMemcpy(C, d_C, size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
}
